#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <errno.h>
#include <chrono>
#include <iostream>
#include "lzlocal.h"
#include "bitfile.h"
#include "matcher_base.h"

#define checkError(ans)                       \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__global__ void FindMatchBatchKernel(char *buffer, int bufferSize, int *matches_length, int *matches_offset, int bufferSizeAdjusted, int currentMatchCount, bool isLast)
{

    int idX = blockIdx.x * blockDim.x + threadIdx.x;
    int i = WINDOW_SIZE + idX;
    int beginSearch = idX;
    if (i >= bufferSizeAdjusted)
    {
        return;
    }

    int length = 0;
    int offset = 0;
    int windowHead = (currentMatchCount + idX) % WINDOW_SIZE;

    int currentOffset = 0;
    // printf("%i",i);
#define optimized
    int j = 0;

#ifdef optimized
    char current[MAX_CODED];
    for (int j = 0; j < MAX_CODED && i + j < bufferSizeAdjusted; j++)
    {
        current[j] = buffer[i + j];
    }

    int qq = WINDOW_SIZE;

    int numSubMat = qq / BLOCK_SIZE;

    __shared__ char bufferCache[WINDOW_SIZE + BLOCK_SIZE];
    const int beginCache = blockIdx.x * blockDim.x;
    for (int k = 0; k < numSubMat; k++)
    {
        if ((beginCache) + BLOCK_SIZE * k + threadIdx.x < bufferSize)
            bufferCache[BLOCK_SIZE * k + threadIdx.x] = buffer[(beginCache) + BLOCK_SIZE * k + threadIdx.x]; //buffer[beginSearch + numSubMat * k + threadIdx.x];
    }
    if (beginCache + WINDOW_SIZE + threadIdx.x < bufferSize)
        bufferCache[WINDOW_SIZE + threadIdx.x] = buffer[beginCache + WINDOW_SIZE + threadIdx.x]; //buffer[beginSearch + numSubMat * k + threadIdx.x];

    __syncthreads();
    const int beginInBufferCache = idX - blockIdx.x * blockDim.x;
#else
    char *current = buffer + i;
#endif
    // const int showThis = 4096 * 2;
    // if(idX == showThis){
    //     printf("AAA %i %i\n", blockIdx.x*blockDim.x - WINDOW_SIZE,idX - blockIdx.x*blockDim.x);
    //     for(int k = 0; k < WINDOW_SIZE ;k++){
    //         printf("%c",bufferCache[beginInBufferCache+ k]);
    //     }
    //     for(int k = 0; k < 10;k++)
    //         printf("\n");

    //     for(int k = 0; k < WINDOW_SIZE;k++){
    //         printf("%c",buffer[beginSearch + k]);
    //     }

    //     for(int k = 0; k < 10;k++)
    //         printf("\n");

    // }

    while (1)
    {
#ifdef optimized
        if (current[0] == bufferCache[beginInBufferCache + Wrap((currentOffset), WINDOW_SIZE)])
#else
        if (current[0] == buffer[beginSearch + Wrap((currentOffset), WINDOW_SIZE)])
#endif
        {
            /* we matched one. how many more match? */
            j = 1;

            while (
#ifdef optimized
                current[j] == bufferCache[beginInBufferCache + Wrap((currentOffset + j), WINDOW_SIZE)]
#else
                current[j] == buffer[beginSearch + Wrap((currentOffset + j), WINDOW_SIZE)]

#endif
                && (!isLast ||
                    (beginSearch + Wrap((currentOffset + j), WINDOW_SIZE) < bufferSizeAdjusted && i + j < bufferSizeAdjusted)))
            {

                if (j >= MAX_CODED)
                {
                    break;
                }
                j++;
            }

            if (j > length)
            {

                length = j;
                offset = Wrap((currentOffset + windowHead), WINDOW_SIZE);
            }
        }

        if (j >= MAX_CODED)
        {
            length = MAX_CODED;
            break;
        }

        currentOffset++;

        if (currentOffset == WINDOW_SIZE)
        {
            break;
        }
    }
    matches_offset[idX] = offset;
    matches_length[idX] = length;
}
int MatcherCuda::Init()
{
    MatcherBase::Init();
    return 0;
}
int MatcherCuda::FindMatchBatch(char *buffer, int bufferSize, int *matches_length, int *matches_offset, int *matchSize, bool isLast, int currentMatchCount)
{
    int bufferSizeAdjusted = bufferSize - MAX_CODED;
    if (isLast)
    {
        bufferSizeAdjusted += MAX_CODED;
    }
    int matchCount = bufferSizeAdjusted - WINDOW_SIZE;
    *matchSize = matchCount;

    int sizeToLaunch = matchCount;
    int blocks = sizeToLaunch / BLOCK_SIZE + (sizeToLaunch % BLOCK_SIZE > 0 ? 1 : 0);
    int threads = BLOCK_SIZE;

    char *d_buffer;
    int *d_matches_length;
    int *d_matches_offset;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float miliseconds;

    hipEventRecord(start, 0);
    //std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    checkError(hipMalloc((void **)&d_buffer, sizeof(char) * bufferSize));
    checkError(hipMalloc((void **)&d_matches_length, sizeof(int) * matchCount));
    checkError(hipMalloc((void **)&d_matches_offset, sizeof(int) * matchCount));

    checkError(hipMemcpy(d_buffer, buffer, sizeof(char) * bufferSize, hipMemcpyHostToDevice));

    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&miliseconds, start, stop);

    timeSpentOnMemoryHostToDevice += miliseconds;
    //std::chrono::steady_clock::time_point end= std::chrono::steady_clock::now();
    //timeSpentOnMemoryHostToDevice += std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();

    //begin = std::chrono::steady_clock::now();
    hipEventRecord(start, 0);
    FindMatchBatchKernel<<<blocks, threads>>>(d_buffer, bufferSize, d_matches_length, d_matches_offset, bufferSizeAdjusted, currentMatchCount, isLast);
    checkError(hipPeekAtLastError());
    //end= std::chrono::steady_clock::now();
    //timeSpentOnKernel += std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&miliseconds, start, stop);
    timeSpentOnKernel += miliseconds;
    // hipDeviceSynchronize();
    //begin = std::chrono::steady_clock::now();
    hipEventRecord(start, 0);
    checkError(hipMemcpy(matches_offset, d_matches_offset, sizeof(int) * matchCount, hipMemcpyDeviceToHost));
    checkError(hipMemcpy(matches_length, d_matches_length, sizeof(int) * matchCount, hipMemcpyDeviceToHost));

    hipFree(d_buffer);
    hipFree(d_matches_length);
    hipFree(d_matches_offset);

    //end= std::chrono::steady_clock::now();
    //timeSpentOnMemoryDeviceToHost += std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&miliseconds, start, stop);
    timeSpentOnMemoryDeviceToHost += miliseconds;
    return 0;
}
