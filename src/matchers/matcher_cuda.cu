#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <errno.h>
#include <chrono>
#include <iostream>
#include "lzlocal.h"
#include "lzss.h"
#include "bitfile.h"
#include "matcher_base.h"
#include "gpu_util.h"
#define checkError(ans)                       \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}
__global__ void FillBuffer(char * buffer, unsigned char * input, int inputSize){
    int idX = blockIdx.x * blockDim.x + threadIdx.x;
    if(idX < WINDOW_SIZE){
        buffer[idX] = ' ';
    }else if(idX - WINDOW_SIZE < inputSize){
        buffer[idX] = input[idX - WINDOW_SIZE];
    }
}
__global__ void FindMatchBatchKernel(char *buffer, int bufferSize, int *matches_length, int *matches_offset, int bufferSizeAdjusted, int currentMatchCount, bool isLast)
{

    int idX = blockIdx.x * blockDim.x + threadIdx.x;
    int i = WINDOW_SIZE + idX;
    int beginSearch = idX;
    if (i >= bufferSizeAdjusted)
    {
        return;
    }

    int length = 0;
    int offset = 0;
    int windowHead = (currentMatchCount + idX) % WINDOW_SIZE;

    int currentOffset = 0;

    //Uncoded Lookahead optimization
    char current[MAX_CODED];
    //for (int j = 0; j < MAX_CODED && i + j < bufferSizeAdjusted; j++)
    for (int j = 0; j < MAX_CODED; j++)
    {
        current[j] = buffer[i + j];
    }
//    char* current = buffer + i;
    int j = 0;
    while (1)
    {
        if (current[0] == buffer[beginSearch + Wrap((currentOffset), WINDOW_SIZE)])
        {
            /* we matched one. how many more match? */
            j = 1;

            while (
              current[j] == buffer[beginSearch + Wrap((currentOffset + j), WINDOW_SIZE)] && (!isLast ||
                                                                                               (beginSearch + Wrap((currentOffset + j), WINDOW_SIZE) < bufferSizeAdjusted && i + j < bufferSizeAdjusted)))
            {

                if (j >= MAX_CODED)
                {
                    break;
                }
                j++;
            }

            if (j > length)
            {

                length = j;
                offset = Wrap((currentOffset + windowHead), WINDOW_SIZE);
            }
        }

        if (j >= MAX_CODED)
        {
            length = MAX_CODED;
            break;
        }

        currentOffset++;

        if (currentOffset == WINDOW_SIZE)
        {
            break;
        }
    }
    matches_offset[idX] = offset;
    matches_length[idX] = length;
}



__global__ void FindMatchBatchKernelWithoutBuffer(unsigned char *buffer, int bufferSize, int *matches_length, int *matches_offset)
{

    int idX = blockIdx.x * blockDim.x + threadIdx.x;
    int i = idX;
    int beginSearch = idX - WINDOW_SIZE;
    if (i >= bufferSize)
    {
        return;
    }

    int length = 0;
    int offset = 0;
    int windowHead = ( idX) % WINDOW_SIZE;

    int currentOffset = 0;

    //Uncoded Lookahead optimization
    char current[MAX_CODED];
    //for (int j = 0; j < MAX_CODED && i + j < bufferSizeAdjusted; j++)
    for (int j = 0; j < MAX_CODED; j++)
    {
        current[j] = buffer[i + j];
    }

    //First WINDOW_SIZE bits will always be ' ', optimize begging where data really is
    if(beginSearch < -MAX_CODED){
        currentOffset = (beginSearch * -1) - MAX_CODED;
    }
//    char* current = buffer + i;
    int j = 0;
    while (1)
    {
        if (current[0] == (beginSearch + Wrap((currentOffset), WINDOW_SIZE) < 0? ' ': buffer[beginSearch + Wrap((currentOffset), WINDOW_SIZE)]))
        {
            /* we matched one. how many more match? */
            j = 1;

            while (
              current[j] == (beginSearch + Wrap((currentOffset + j), WINDOW_SIZE) < 0?' ':buffer[beginSearch + Wrap((currentOffset + j), WINDOW_SIZE)]) &&  
                beginSearch + Wrap((currentOffset + j), WINDOW_SIZE) < bufferSize && i + j < bufferSize)
            {

                if (j >= MAX_CODED)
                {
                    break;
                }
                j++;
            }

            if (j > length)
            {

                length = j;
                offset = Wrap((currentOffset + windowHead), WINDOW_SIZE);
            }
        }

        if (j >= MAX_CODED)
        {
            length = MAX_CODED;
            break;
        }
        
        currentOffset++;

        if (currentOffset == WINDOW_SIZE)
        {
            break;
        }
    }
    matches_offset[idX] = offset;
    matches_length[idX] = length;
}
int MatcherCuda::Init()
{
    MatcherBase::Init();
    return 0;
}
int MatcherCuda::FindMatchBatch(char *buffer, int bufferSize, int *matches_length, int *matches_offset, int *matchSize, bool isLast, int currentMatchCount, int currentBatch)
{
    auto deviceIds = getDeviceIds();
    int deviceIdThread = deviceIds[currentBatch % deviceIds.size()];
    hipSetDevice(deviceIdThread);
    int bufferSizeAdjusted = bufferSize - MAX_CODED;
    if (isLast)
    {
        bufferSizeAdjusted += MAX_CODED;
    }
    int matchCount = bufferSizeAdjusted - WINDOW_SIZE;
    *matchSize = matchCount;

    int sizeToLaunch = matchCount;
    int blocks = sizeToLaunch / BLOCK_SIZE + (sizeToLaunch % BLOCK_SIZE > 0 ? 1 : 0);
    int threads = BLOCK_SIZE;

    char *d_buffer;
    int *d_matches_length;
    int *d_matches_offset;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float miliseconds;

    hipEventRecord(start, 0);
    //std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    checkError(hipMalloc((void **)&d_buffer, sizeof(char) * bufferSize));
    checkError(hipMalloc((void **)&d_matches_length, sizeof(int) * matchCount));
    checkError(hipMalloc((void **)&d_matches_offset, sizeof(int) * matchCount));

    checkError(hipMemcpy(d_buffer, buffer, sizeof(char) * bufferSize, hipMemcpyHostToDevice));

    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&miliseconds, start, stop);

    timeSpentOnMemoryHostToDevice += miliseconds;
    //std::chrono::steady_clock::time_point end= std::chrono::steady_clock::now();
    //timeSpentOnMemoryHostToDevice += std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();

    //begin = std::chrono::steady_clock::now();
    hipEventRecord(start, 0);
    FindMatchBatchKernel<<<blocks, threads>>>(d_buffer, bufferSize, d_matches_length, d_matches_offset, bufferSizeAdjusted, currentMatchCount, isLast);
    checkError(hipPeekAtLastError());
    //end= std::chrono::steady_clock::now();
    //timeSpentOnKernel += std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&miliseconds, start, stop);
    timeSpentOnKernel += miliseconds;
    // hipDeviceSynchronize();
    //begin = std::chrono::steady_clock::now();
    hipEventRecord(start, 0);
    checkError(hipMemcpy(matches_offset, d_matches_offset, sizeof(int) * matchCount, hipMemcpyDeviceToHost));
    checkError(hipMemcpy(matches_length, d_matches_length, sizeof(int) * matchCount, hipMemcpyDeviceToHost));

    hipFree(d_buffer);
    hipFree(d_matches_length);
    hipFree(d_matches_offset);

    //end= std::chrono::steady_clock::now();
    //timeSpentOnMemoryDeviceToHost += std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&miliseconds, start, stop);
    timeSpentOnMemoryDeviceToHost += miliseconds;
    return 0;
}




int MatcherCuda::FindMatchBatchUsingDeviceInput(unsigned char *input, int inputSize, int *matches_length, int *matches_offset, int *matchSize, int deviceIndex)
{
    // auto deviceIds = getDeviceIds();
    // int deviceIdThread = deviceIds[deviceIndex];
    hipSetDevice(deviceIndex);

    int bufferSizeAdjusted = inputSize + WINDOW_SIZE;

    int matchCount = inputSize;
    *matchSize = inputSize;

    
    // char *d_buffer;
    int *d_matches_length;
    int *d_matches_offset;

    int sizeToLaunch ;
    int blocks;
    int threads;
    

    // checkError(hipMalloc((void **)&d_buffer, sizeof(char) * (inputSize + WINDOW_SIZE)));

    // int sizeToLaunch = bufferSizeAdjusted;
    // int blocks = sizeToLaunch / BLOCK_SIZE + (sizeToLaunch % BLOCK_SIZE > 0 ? 1 : 0);
    // int threads = BLOCK_SIZE;
    // FillBuffer<<<blocks,threads>>>(d_buffer,input,inputSize);
    
    checkError(hipMalloc((void **)&d_matches_length, sizeof(int) * matchCount));
    checkError(hipMalloc((void **)&d_matches_offset, sizeof(int) * matchCount));

    sizeToLaunch = matchCount;
    blocks = sizeToLaunch / BLOCK_SIZE + (sizeToLaunch % BLOCK_SIZE > 0 ? 1 : 0);
    threads = BLOCK_SIZE;

    FindMatchBatchKernelWithoutBuffer<<<blocks, threads>>>(input, inputSize, d_matches_length, d_matches_offset);
    // FindMatchBatchKernel<<<blocks, threads>>>(d_buffer, bufferSizeAdjusted, d_matches_length, d_matches_offset, bufferSizeAdjusted, 0, true);
    checkError(hipPeekAtLastError());

    checkError(hipMemcpy(matches_offset, d_matches_offset, sizeof(int) * matchCount, hipMemcpyDeviceToHost));
    checkError(hipMemcpy(matches_length, d_matches_length, sizeof(int) * matchCount, hipMemcpyDeviceToHost));

    // hipFree(d_buffer);
    hipFree(d_matches_length);
    hipFree(d_matches_offset);

    return 0;
}

